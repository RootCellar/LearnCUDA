#include "hip/hip_runtime.h"
/*
 *
 * Darian Marvel
 * 3/06/2023
 * 
 * Benchmarking different Nvidia graphics card operations
 *
 *
*/



// Includes

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <stdbool.h>

#include "debug.h"

#define SECONDS_PER_RUN 1
#define BLOCKS 128*1000

/*
 *
 * BENCHMARK(function, values, "bench") will expand to the following:
 *

    do {
        int runCount = 0;
        clock_t start_time = clock();
        clock_t time_now;
        while( (time_now = clock() ) - start_time < CLOCKS_PER_SEC * SECONDS_PER_RUN) {
            function<<<BLOCKS/128, 128>>>(values);
            runCount++;
        }

        float seconds = (float) (time_now - start_time) / CLOCKS_PER_SEC;
        debug_printf("bench: %d over %f seconds\n", runCount, seconds);
    } while(0)

 *
 * This makes benchmarking much easier. The do-while loop surrounds the whole thing just make sure
 * it is in it's own code block and won't behave weirdly inside of other blocks (for loops, etc)
 *
*/

#define BENCHMARK(x, vals, name) do { int runCount = 0; clock_t start_time = clock(); clock_t time_now;\
    while( ( time_now = clock() ) - start_time < CLOCKS_PER_SEC * SECONDS_PER_RUN) { x<<<BLOCKS/128, 128>>>(vals); runCount++; }\
    float seconds = (float) (time_now - start_time) / CLOCKS_PER_SEC; debug_printf(name ": %d over %f seconds\n", runCount, seconds); } while(0)

__global__ void benchFloats(float* floats) {

    int j = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i = 0; i < 1000; i++) {
        floats[j] *= 6.7;
        floats[j] *= 6.7;
        floats[j] *= 6.7;
        floats[j] *= 6.7;
        floats[j] *= 6.7;
        floats[j] *= 6.7;
        floats[j] *= 6.7;
        floats[j] *= 6.7;
        floats[j] *= 6.7;
        floats[j] *= 6.7;
    }

}

/*
void benchmark(void* func, void* values, char* name) {
    int runCount;
    clock_t start_time, time_now;
    float seconds;

    runCount = 0;
    start_time = clock();
    while( (time_now = clock() ) - start_time < CLOCKS_PER_SEC * SECONDS_PER_RUN) {
        (**func)<<<BLOCKS/128, 128>>>(values);
        runCount++;
    }

    seconds = (float) (time_now - start_time) / CLOCKS_PER_SEC;
    debug_printf("benchFloats: %d over %f seconds\n", runCount, seconds);
}
*/

int main(int argc, char** argv) {
    clock_t start_time, time_now;
    int runCount;
    float seconds;

    float* floats = (float*) malloc(sizeof(float) * BLOCKS);
    float* gpu_floats;

    hipMalloc(&gpu_floats, sizeof(float) * BLOCKS);

    if(gpu_floats == 0 || floats == 0) {
        perror("Could not allocate space for floats\n");
        exit(1);
    }

    for(int i = 0; i < BLOCKS; i++) {
        floats[i] = (float) i;
    }

    runCount = 0;
    start_time = clock();
    while( (time_now = clock() ) - start_time < CLOCKS_PER_SEC * SECONDS_PER_RUN) {
        hipMemcpy(gpu_floats, floats, sizeof(float) * BLOCKS, hipMemcpyHostToDevice);
        runCount++;
    }

    seconds = (float) (time_now - start_time) / CLOCKS_PER_SEC;
    debug_printf("hipMemcpy: %d over %f seconds\n", runCount, seconds);

    debug_printf("Copied %ld bytes\n", sizeof(float) * BLOCKS);

    /*
    runCount = 0;
    start_time = clock();
    while( (time_now = clock() ) - start_time < CLOCKS_PER_SEC * SECONDS_PER_RUN) {
        benchFloats<<<BLOCKS/128, 128>>>(gpu_floats);
        runCount++;
    }

    seconds = (float) (time_now - start_time) / CLOCKS_PER_SEC;
    debug_printf("benchFloats: %d over %f seconds\n", runCount, seconds);
    */

    for(int i = 0; i < 10; i++) {
        //hipMemcpy(gpu_floats, floats, sizeof(float) * BLOCKS, hipMemcpyHostToDevice);
        BENCHMARK(benchFloats, gpu_floats, "benchFloats");
    }

}