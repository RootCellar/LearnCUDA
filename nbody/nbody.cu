#include "hip/hip_runtime.h"
/*
 *
 * Darian Marvel
 * 2/24/2023
 * 
 * Building an n-particle/n-body simulator that runs on the graphics card
 *
 * Uses some code from Ryan Brune's Orbital-Simulator
 *
*/



// Includes

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <GL/gl.h>
#include <GL/glut.h>
#include <time.h>
#include <GL/glu.h>
#include <unistd.h>
#include <stdbool.h>

#include "debug.h"
#include "physics.h"

#define WIDTH 1600
#define HEIGHT 800

#define PI 3.14159265359

void DrawCircle(float cx, float cy, float r) {
	glBegin(GL_LINE_LOOP);
	for(float i = 0; i < 2 * PI ; i += 0.1)
	{
		float x = r * cosf(i);
        x /= WIDTH/HEIGHT;

		float y = r * sinf(i);
		glVertex2f(x + cx, y + cy);
	}
	glEnd();
}

__device__ void calcDistance(float* distance, struct particle one, struct particle two) {
    (*distance) = sqrtf( pow(one.x-two.x,2) + pow(one.y-two.y,2) );
}

__global__ void calcAcceleration(struct particle* particles, struct particle center_of_mass) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    //particles[j].v_x = 1;
    //particles[j].v_y = 1;

    float distance;
    calcDistance(&distance, particles[j], center_of_mass);

    if(distance < 1) distance = 1;
    
    float force = 1 / powf(distance, 2);
    force /= 10;

    float x_distance = particles[j].x - center_of_mass.x;
    float y_distance = particles[j].y - center_of_mass.y;

    float angle = atanf(y_distance/x_distance);

    float v_x_add = cosf(angle) * force;
    float v_y_add = sinf(angle) * force;

    if(particles[j].x > center_of_mass.x) {
        v_x_add *= -1;
        v_y_add *= -1;
    }

    /*
    if(particles[j].y > center_of_mass.y) {
        v_y_add *= -1;
    }
    */

    particles[j].v_x += v_x_add;
    particles[j].v_y += v_y_add;


    particles[j].x += particles[j].v_x;
    particles[j].y += particles[j].v_y;
}

int main( int argc, char** argv) {
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_SINGLE);    // Use single color buffer and no depth buffer.
    glutInitWindowSize(WIDTH,HEIGHT);         // Size of display area, in pixels.
    glutInitWindowPosition(0,0);     // Location of window in screen coordinates.
    glutCreateWindow("N-Particle Simulator"); // Parameter is window title.

    int particle_count = 128*20;
    struct particle* particles = (struct particle*) malloc(sizeof(particle) * particle_count);
    if(particles == 0) {
        printf("Could not allocate memory for particles!\n");
        exit(1);
    }

    struct particle* gpu_particles;
    hipMalloc(&gpu_particles, sizeof(struct particle) * particle_count);
    if(gpu_particles == 0) {
        printf("Could not allocate GPU memory for particles!\n");
        exit(1);
    }

    struct particle center_of_mass;
    float scaled_x;
    float scaled_y;

    // Position Particles
    for(int i = 0; i < particle_count; i++) {
        particles[i].x = i % 100;
        particles[i].x *= 15;
        particles[i].y = i / 100;
        particles[i].y *= 30;

        particles[i].v_x = 0;
        particles[i].v_y = 0;
    }

    while(1) {

        // Do the physics

        calculate_center_of_mass(&center_of_mass, particles, particle_count);

        hipMemcpy(gpu_particles, particles, particle_count * sizeof(struct particle), hipMemcpyHostToDevice);

        calcAcceleration<<<particle_count/128,128>>>(gpu_particles, center_of_mass);

        hipMemcpy(particles, gpu_particles, particle_count * sizeof(struct particle), hipMemcpyDeviceToHost);

        // Draw it

        glClear(GL_COLOR_BUFFER_BIT);

        scaled_x = (center_of_mass.x - WIDTH/2) / (WIDTH/2);
        scaled_y = (center_of_mass.y - HEIGHT/2) / (HEIGHT/2);
        DrawCircle(scaled_x, scaled_y, 0.01);

        for(int i = 0; i < particle_count; i++) {
            scaled_x = (particles[i].x - WIDTH/2) / (WIDTH/2);
            scaled_y = (particles[i].y - HEIGHT/2) / (HEIGHT/2);
            DrawCircle(scaled_x, scaled_y, 0.0025);
        }

        glutSwapBuffers();

        usleep(1000);
    }
}