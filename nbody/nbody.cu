#include "hip/hip_runtime.h"
/*
 *
 * Darian Marvel
 * 2/24/2023
 * 
 * Building an n-particle/n-body simulator that runs on the graphics card
 *
 * Uses some code from Ryan Brune's Orbital-Simulator
 *
*/



// Includes

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <GL/gl.h>
#include <GL/glut.h>
#include <time.h>
#include <GL/glu.h>
#include <unistd.h>
#include <stdbool.h>

#include "debug.h"
#include "physics.h"

#define WIDTH 1600
#define HEIGHT 800

#define PI 3.14159265359

void DrawCircle(float cx, float cy, float r) {
	glBegin(GL_LINE_LOOP);
	for(float i = 0; i < 2 * PI ; i += 0.1)
	{
		float x = r * cosf(i);
        x /= WIDTH/HEIGHT;

		float y = r * sinf(i);
		glVertex2f(x + cx, y + cy);
	}
	glEnd();
}

__device__ void calcDistance(float* distance, struct particle one, struct particle two) {
    (*distance) = sqrtf( pow(one.x-two.x,2) + pow(one.y-two.y,2) );
}

__global__ void calcAcceleration(struct particle* particles, struct particle center_of_mass) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    particles[j].v_x = 1;
    particles[j].v_y = 1;

    float distance;
    calcDistance(&distance, particles[j], center_of_mass);

    particles[j].x += particles[j].v_x;
    particles[j].y += particles[j].v_y;
}

int main( int argc, char** argv) {
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_SINGLE);    // Use single color buffer and no depth buffer.
    glutInitWindowSize(WIDTH,HEIGHT);         // Size of display area, in pixels.
    glutInitWindowPosition(0,0);     // Location of window in screen coordinates.
    glutCreateWindow("N-Particle Simulator"); // Parameter is window title.

    int particle_count = 128*20;
    struct particle* particles = (struct particle*) malloc(sizeof(particle) * particle_count);
    if(particles == 0) {
        printf("Could not allocate memory for particles!\n");
        exit(1);
    }

    struct particle* gpu_particles;
    hipMalloc(&gpu_particles, sizeof(struct particle) * particle_count);
    if(gpu_particles == 0) {
        printf("Could not allocate GPU memory for particles!\n");
        exit(1);
    }

    struct particle center_of_mass;
    float scaled_x;
    float scaled_y;

    // Position Particles
    for(int i = 0; i < particle_count; i++) {
        particles[i].x = i % 100;
        particles[i].x *= 15;
        particles[i].y = i / 100;
        particles[i].y *= 30;
    }

    while(1) {

        // Do the physics

        calculate_center_of_mass(&center_of_mass, particles, particle_count);

        hipMemcpy(gpu_particles, particles, particle_count * sizeof(struct particle), hipMemcpyHostToDevice);

        calcAcceleration<<<particle_count/128,128>>>(gpu_particles, center_of_mass);

        hipMemcpy(particles, gpu_particles, particle_count * sizeof(struct particle), hipMemcpyDeviceToHost);

        // Draw it

        glClear(GL_COLOR_BUFFER_BIT);
        //DrawCircle(-0.8, 0.1, 0.01);

        for(int i = 0; i < particle_count; i++) {
            scaled_x = (particles[i].x - WIDTH/2) / (WIDTH/2);
            scaled_y = (particles[i].y - HEIGHT/2) / (HEIGHT/2);
            DrawCircle(scaled_x, scaled_y, 0.0025);
        }

        glutSwapBuffers();

        usleep(1000);
    }
}