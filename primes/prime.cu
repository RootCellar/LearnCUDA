/*
 * RootCellar
 * 10/17/2022
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define DEBUG 1

#define debug_print(fmt, ...) \
        do { if (DEBUG) fprintf(stdout, "%s:%d:%s(): " fmt, __FILE__, \
                                __LINE__, __func__, __VA_ARGS__); } while (0)

__global__
void isPrime(int n, float a, int *x, int *nums)
{
  // Find which number we are checking
  int j = blockIdx.x*blockDim.x + threadIdx.x;

  // Assume it is prime
  x[j]=1;

  if(nums[j] < 2) {
    x[j] = 0;
    return;
  }

  // Find a case that means it isn't prime
  for(int i=2; i <= sqrtf(nums[j]); i++) {
        if(nums[j]%i == 0) {
          x[j]=0;
          return;
        }
  }

}

// TODO: for large number of primes, cycle through sets of them at a set size
// so that any number of primes can be found (assuming large hard drive space)
// on any RAM or VRAM size

// PSEUDOCODE:

/*

  for(each set)
    setup array in RAM
    copy array to VRAM
    find the primes
    copy back to RAM
    write to file

*/

int main(void)
{
  // Just over half a billion primes at 1<<29, consumes ~2.5 GB VRAM
  // if done in one pass
  int N_total = 1<<30; // the number we will search up to
  int N = N_total/(1<<4); // how many per pass
  int previous_max = 0;

  debug_print("%d primes total, %d primes per pass\n", N_total, N);

  // Pointers
  int *x, *d_x;
  int *nums, *gpu_nums;

  // List of primes in RAM
  x = (int*)malloc(N*sizeof(int));
  nums = (int*) malloc(N*sizeof(int));

  // Same list in VRAM
  hipMalloc(&d_x, N*sizeof(int));
  hipMalloc(&gpu_nums, N*sizeof(int));

  while(previous_max < N_total) {

    debug_print("Handling %d to %d\n", previous_max, previous_max + N);

    /*

    // initialize list
    for (int i = 0; i < N; i++) {
      x[i] = 1;
      nums[i] = i+previous_max;
    }

    // Copy host list to VRAM list
    cudaMemcpy(d_x, x, N*sizeof(int), cudaMemcpyHostToDevice);
    cudaMemcpy(gpu_nums, nums, N*sizeof(int), cudaMemcpyHostToDevice);

    // Run the calculation
    isPrime<<<N/128, 128>>>(N, 2.0f, d_x, gpu_nums);

    // Copy results back to host RAM
    cudaMemcpy(x, d_x, N*sizeof(int), cudaMemcpyDeviceToHost);
    cudaMemcpy(nums, gpu_nums, N*sizeof(int), cudaMemcpyDeviceToHost);

    // Display results
    for(int i=0; i < N; i++) {
      if(x[i] == 1) printf("%d\n", i);
    }

    */

    previous_max = previous_max + N;

  }

  // Cleanup
  hipFree(d_x);
  hipFree(gpu_nums);
  free(x);
  free(nums);
}
